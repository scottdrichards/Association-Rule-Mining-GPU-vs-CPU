#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include "kernel_transpose.h"
#include "kernel_EClen2.h"

__host__ int main()  {
/***************  READING DATABASE FILE  **************************/
    int i,j,k;
    int num_of_itemsets, num_of_transactions;

    FILE *fp;
    fp = fopen("database.txt","r");
    fscanf(fp,"%d",&num_of_transactions);
    fscanf(fp,"%d",&num_of_itemsets);

    char *db[num_of_transactions][num_of_itemsets];
    for (i = 0; i < num_of_transactions; i++)  {
        for (j = 0; j < num_of_itemsets; j++)  {
            db[i][j] = (char *) malloc (100 * sizeof(char));
            memcpy(db[i][j],"",100 * sizeof(char));
        }
    }

    i = 0;
    j = 0;
    const char delim[20] = "\n ,{}:";
    char *token;
    int approx_size = (num_of_itemsets+2) * 100 * sizeof(char);
    char dummy[approx_size];
    memset(dummy,0,approx_size);

    while(fgets(dummy,approx_size,fp) != NULL)  {
	token = strtok(dummy,delim);
	if (token == NULL)  continue;   // To flush out the termination null char
	token = strtok(NULL,delim);
	while (token != NULL)  {
	    memcpy(db[i][j],token,100 * sizeof(char));
	    token = strtok(NULL,delim);
	    j++;
	}
	j = 0;
	i++;
	memset(dummy,0,approx_size);
    }		

    printf("++++++++++  Database  ++++++++++\n");
    for (i = 0; i < num_of_transactions; i++)  {
        for (j = 0; j < num_of_itemsets; j++)  {
            printf("%s ",db[i][j]);
        }
        printf("\n");
    }

    fclose(fp);
/*******************  END  ***********************/

/*******************  OTHER INITIALIZATIONS  ********************************/
    char *db_tran[num_of_itemsets*num_of_transactions][num_of_transactions+1];
    for (i = 0; i < num_of_itemsets*num_of_transactions; i++)  {
        for (j = 0; j < num_of_transactions+1; j++)  {
            db_tran[i][j] = (char*) malloc (100 * sizeof(char));
            memcpy(db_tran[i][j],"",100 * sizeof(char));
        }
    }

    int EC_size = num_of_itemsets * (num_of_itemsets - 1) / 2;
    char *EC_len2[num_of_transactions*EC_size][(num_of_transactions+2)*27];
    for (i = 0; i < num_of_transactions*EC_size; i++)  {
	for (j = 0; j < (num_of_transactions+2)*27; j++)  {
	    EC_len2[i][j] = (char*) malloc (100 * sizeof(char));
            memcpy(EC_len2[i][j],"",100 * sizeof(char));
	}
    }
/*******************  END  ***********************/

/*******************  CUDA HOST TO DEVICE COPY  ***************************************/
    dim3 db_gpuDim(100,num_of_transactions,num_of_itemsets);
    hipExtent db_gpuVolSizeBytes = make_hipExtent(sizeof(char) * db_gpuDim.x, db_gpuDim.y, db_gpuDim.z);
    hipPitchedPtr db_gpuPitchPtr;
    hipMalloc3D(&db_gpuPitchPtr,db_gpuVolSizeBytes);
    hipMemset3D(db_gpuPitchPtr,0,db_gpuVolSizeBytes);

    dim3 db_tran_gpuDim(100,num_of_itemsets*num_of_transactions,num_of_transactions+1);
    hipExtent db_tran_gpuVolSizeBytes = make_hipExtent(sizeof(char) * db_tran_gpuDim.x, db_tran_gpuDim.y, db_tran_gpuDim.z);
    hipPitchedPtr db_tran_gpuPitchPtr;
    hipMalloc3D(&db_tran_gpuPitchPtr,db_tran_gpuVolSizeBytes);
    hipMemset3D(db_tran_gpuPitchPtr,0,db_tran_gpuVolSizeBytes);

    dim3 EC_len2_gpuDim(100,num_of_transactions*EC_size,(num_of_transactions+2)*27);
    hipExtent EC_len2_gpuVolSizeBytes = make_hipExtent(sizeof(char) * EC_len2_gpuDim.x, EC_len2_gpuDim.y, EC_len2_gpuDim.z);
    hipPitchedPtr EC_len2_gpuPitchPtr;
    hipMalloc3D(&EC_len2_gpuPitchPtr,EC_len2_gpuVolSizeBytes);
    hipMemset3D(EC_len2_gpuPitchPtr,0,EC_len2_gpuVolSizeBytes);

    char *db_ptr = (char *)db_gpuPitchPtr.ptr;
    size_t db_pitch = db_gpuPitchPtr.pitch;
    size_t db_slicePitch = db_pitch * num_of_transactions;
    char *db_current_slice;
    char *db_element;

    for (i = 0; i < num_of_transactions; i++)  {
        for (j = 0; j < num_of_itemsets; j++)  {
	    db_current_slice = db_ptr + j * db_slicePitch;
	    db_element = (char *)(db_current_slice + i * db_pitch);
	    hipMemcpy(db_element,db[i][j],100*sizeof(char),hipMemcpyHostToDevice); 
	}
    }
/*******************  END  *****************************/

/*******************  CUDA KERNEL LAUNCH  *************************************/
    dim3 dimGrid(ceil(sqrt(num_of_transactions)),ceil(sqrt(num_of_transactions)),1); 
    dim3 dimBlock(ceil(sqrt(num_of_itemsets)),ceil(sqrt(num_of_itemsets)),1);

    // Launch kernel to find the transpose of db
    cudaTranspose <<< dimGrid, dimBlock >>> (db_gpuPitchPtr, db_gpuDim, db_tran_gpuPitchPtr);
    hipDeviceSynchronize();

    // Launch kernel to find all the entries of the Equivalent Classes of upto length 2 and sort them simultaneously
    cudaEClen2Part1 <<< dimGrid, dimBlock >>> (db_gpuPitchPtr, db_gpuDim, db_tran_gpuPitchPtr,EC_len2_gpuPitchPtr, EC_size);
    hipDeviceSynchronize();
/*******************  END  *****************************/

/*******************  CUDA DEVICE TO HOST COPY  ********************************************************/
    char *db_tran_ptr = (char*)db_tran_gpuPitchPtr.ptr;
    size_t db_tran_pitch = db_tran_gpuPitchPtr.pitch;
    size_t db_tran_slicePitch = db_tran_pitch * num_of_itemsets * num_of_transactions;
    char *db_tran_current_slice;
    char *db_tran_element;

    for (i = 0; i < num_of_itemsets*num_of_transactions; i++)  {
        for (j = 0; j < num_of_transactions+1; j++)  {
            db_tran_current_slice = db_tran_ptr + j * db_tran_slicePitch;
            db_tran_element = (char *)(db_tran_current_slice + i * db_tran_pitch);
            hipMemcpy(db_tran[i][j],db_tran_element,100*sizeof(char),hipMemcpyDeviceToHost);
        }
    }

    printf("++++++++++  Transpose of the Database  ++++++++++\n");
    for (i = 0; i < num_of_itemsets*num_of_transactions; i++)  {
	if (strcmp(db_tran[i][0],"") == 0)  continue;
	printf("%s: ",db_tran[i][0]);
        for (j = 1; j < num_of_transactions+1; j++)  {
            printf("-%s-",db_tran[i][j]);
        }
        printf("\n");
    }

    char *EC_len2_ptr = (char *)EC_len2_gpuPitchPtr.ptr;
    size_t EC_len2_pitch = EC_len2_gpuPitchPtr.pitch;
    size_t EC_len2_slicePitch = EC_len2_pitch * num_of_transactions * EC_size;
    char *EC_len2_slice;
    char *EC_len2_element;

    for (i = 0; i < num_of_transactions*EC_size; i++)  {
	for (j = 0; j < (num_of_transactions+2)*27; j++)  {
	    EC_len2_slice = EC_len2_ptr + i * EC_len2_pitch;
	    EC_len2_element = (char *)(EC_len2_slice + j * EC_len2_slicePitch);
	    hipMemcpy(EC_len2[i][j],EC_len2_element,100*sizeof(char),hipMemcpyDeviceToHost);
	}
    }

    j = 0;
    printf("++++++++++  Equivalent Class of upto length 2  ++++++++++\n");
    for (k = 0; k < 27; k++)  {
        for (i = 0; i < num_of_transactions*EC_size; i++)  {
            if (strcmp(EC_len2[i][(k*(num_of_transactions+2))+0],"") == 0)  continue;
	    printf("%s%s: ",EC_len2[i][(k*(num_of_transactions+2))+0],EC_len2[i][(k*(num_of_transactions+2))+1]);
            for (j = 2; j < num_of_transactions+2; j++)  {
                printf("-%s-",EC_len2[i][(k*(num_of_transactions+2))+j]);
            }
            printf("\n");
	}
    }
/*******************  END  *****************************/
    printf("++++++++++  End  ++++++++++\n");

    hipFree(db_gpuPitchPtr.ptr);
    hipFree(db_tran_gpuPitchPtr.ptr);

//  free(db);
    return 0;
}

