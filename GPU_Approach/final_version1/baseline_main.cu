#include "hip/hip_runtime.h"
/*********************  ASSUMPTIONS  ******************************
1) num_of_itemsets is the Upper limit of num of items per transaction; 
   i.e. num of items per transaction <= num_of_itemsets
2) string length = 100 characters max
*********************  END  ******************************/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include "kernel_transpose.h"
#include "kernel_EClen2.h"

__host__ int main()  {
/***************  READING DATABASE FILE  **************************/
    int i,j;
    int num_of_itemsets, num_of_transactions;

    FILE *fp;
    fp = fopen("database.txt","r");
    fscanf(fp,"%d",&num_of_transactions);
    fscanf(fp,"%d",&num_of_itemsets);

    char *db[num_of_transactions][num_of_itemsets];
    for (i = 0; i < num_of_transactions; i++)  {
        for (j = 0; j < num_of_itemsets; j++)  {
            db[i][j] = (char *) malloc (100 * sizeof(char));
            memcpy(db[i][j],"",100 * sizeof(char));
        }
    }

    i = 0;
    j = 0;
    const char delim[20] = "\n ,{}:";
    char *token;
    int approx_size = (num_of_itemsets+2) * 100 * sizeof(char);  // +2 is due to delimeters , { } : etc. Size was 1<n<2, so I took 2
    char dummy[approx_size];
    memset(dummy,0,approx_size);

    while(fgets(dummy,approx_size,fp) != NULL)  {
	token = strtok(dummy,delim);
	if (token == NULL)  continue;   // To flush out the termination null char
	token = strtok(NULL,delim);
	while (token != NULL)  {
	    memcpy(db[i][j],token,100 * sizeof(char));
	    token = strtok(NULL,delim);
	    j++;
	}
	j = 0;
	i++;
	memset(dummy,0,approx_size);
    }		

    printf("++++++++++  Database  ++++++++++\n");
    for (i = 0; i < num_of_transactions; i++)  {
        for (j = 0; j < num_of_itemsets; j++)  {
            printf("%s ",db[i][j]);
        }
        printf("\n");
    }

    fclose(fp);
/*******************  END  ***********************/

/*******************  OTHER INITIALIZATIONS  ********************************/
    char *db_tran[num_of_itemsets*num_of_transactions][num_of_transactions+1];
    // There can be atmost num_of_itemsets*num_of_transactions different elements - HIGH SKEW db case
    // Column 0 is dedicated for elements, column 1 to num_of_transactions+1 are dedicated for attendance

    for (i = 0; i < num_of_itemsets*num_of_transactions; i++)  {
        for (j = 0; j < num_of_transactions+1; j++)  {
            db_tran[i][j] = (char*) malloc (100 * sizeof(char));
            memcpy(db_tran[i][j],"",100 * sizeof(char));
        }
    }

    int EC_size = num_of_itemsets * (num_of_itemsets - 1) / 2;
/*************  Redundant  *****************
    char *EC_len2[num_of_transactions*EC_size][(num_of_transactions+2)*27];

    for (i = 0; i < num_of_transactions*EC_size; i++)  {
	for (j = 0; j < (num_of_transactions+2)*27; j++)  {
	    EC_len2[i][j] = (char*) malloc (100 * sizeof(char));
            memcpy(EC_len2[i][j],"",100 * sizeof(char));
	}
    }
*************  End  ******************/
    char *output_element_len2[num_of_transactions*EC_size*2][2];  // say (A,B) then o/p is A => B and B => A, so *2 num of rows
    for (i = 0; i < num_of_transactions*EC_size*2; i++)  {
        for (j = 0; j < 2; j++)  {
            output_element_len2[i][j] = (char*) malloc (100 * sizeof(char));
            memcpy(output_element_len2[i][j],"",100 * sizeof(char));
        }
    }
/*******************  END  ***********************/

/*******************  CUDA HOST TO DEVICE COPY  ***************************************/
    dim3 db_gpuDim(100,num_of_transactions,num_of_itemsets);
    hipExtent db_gpuVolSizeBytes = make_hipExtent(sizeof(char) * db_gpuDim.x, db_gpuDim.y, db_gpuDim.z);
    hipPitchedPtr db_gpuPitchPtr;
    hipMalloc3D(&db_gpuPitchPtr,db_gpuVolSizeBytes);
    hipMemset3D(db_gpuPitchPtr,0,db_gpuVolSizeBytes);

    dim3 db_tran_gpuDim(100,num_of_itemsets*num_of_transactions,num_of_transactions+1);
    hipExtent db_tran_gpuVolSizeBytes = make_hipExtent(sizeof(char) * db_tran_gpuDim.x, db_tran_gpuDim.y, db_tran_gpuDim.z);
    hipPitchedPtr db_tran_gpuPitchPtr;
    hipMalloc3D(&db_tran_gpuPitchPtr,db_tran_gpuVolSizeBytes);
    hipMemset3D(db_tran_gpuPitchPtr,0,db_tran_gpuVolSizeBytes);

    dim3 EC_len2_gpuDim(100,num_of_transactions*EC_size,(num_of_transactions+2)*27);
    hipExtent EC_len2_gpuVolSizeBytes = make_hipExtent(sizeof(char) * EC_len2_gpuDim.x, EC_len2_gpuDim.y, EC_len2_gpuDim.z);
    hipPitchedPtr EC_len2_gpuPitchPtr;
    hipMalloc3D(&EC_len2_gpuPitchPtr,EC_len2_gpuVolSizeBytes);
    hipMemset3D(EC_len2_gpuPitchPtr,0,EC_len2_gpuVolSizeBytes);

    // There can be atmost num_of_transactions*EC_size different length=2 elements - HIGH SKEW db case
    // Column 0 & 1 are dedicated for the elements,
    // column 2 to num_of_transactions+2 are dedicated for attendance

    /*******************  SPACE vs (PERFORMANCE & CODING SIMPLICITY) TRADE-OFF  *************************
        I chose performance and coding simplicity over space
        space*27 means 27 different sets - 26 sets for english alphabets, 1 set for non-alphabets which include numbers also

        1) Implementing a sorting algorithm in GPU is very complicated
        2) Fastest available sorting algorithm takes O(nlog n) time complexity
        Solution --> Using 27 sets will not only take O(const) time complexity but also make coding simpler
    ********************  END  *********************/

    dim3 output_element_len2_gpuDim(100,num_of_transactions*EC_size*2,2);
    hipExtent output_element_len2_gpuVolSizeBytes = make_hipExtent(sizeof(char) * output_element_len2_gpuDim.x, output_element_len2_gpuDim.y, output_element_len2_gpuDim.z);
    hipPitchedPtr output_element_len2_gpuPitchPtr;
    hipMalloc3D(&output_element_len2_gpuPitchPtr,output_element_len2_gpuVolSizeBytes);
    hipMemset3D(output_element_len2_gpuPitchPtr,0,output_element_len2_gpuVolSizeBytes);

    dim3 output_support_confidence_len2_gpuDim(1,num_of_transactions*EC_size*2,8);
    hipExtent output_support_confidence_len2_gpuVolSizeBytes = make_hipExtent(sizeof(float) * output_support_confidence_len2_gpuDim.x, output_support_confidence_len2_gpuDim.y, output_support_confidence_len2_gpuDim.z);
    hipPitchedPtr output_support_confidence_len2_gpuPitchPtr;
    hipMalloc3D(&output_support_confidence_len2_gpuPitchPtr,output_support_confidence_len2_gpuVolSizeBytes);
    hipMemset3D(output_support_confidence_len2_gpuPitchPtr,0.0000,output_support_confidence_len2_gpuVolSizeBytes);

    char *db_ptr = (char *)db_gpuPitchPtr.ptr;
    size_t db_pitch = db_gpuPitchPtr.pitch;
    size_t db_slicePitch = db_pitch * num_of_transactions;
    char *db_current_slice;
    char *db_element;

    for (i = 0; i < num_of_transactions; i++)  {
        for (j = 0; j < num_of_itemsets; j++)  {
	    db_current_slice = db_ptr + j * db_slicePitch;
	    db_element = (char *)(db_current_slice + i * db_pitch);
	    hipMemcpy(db_element,db[i][j],100*sizeof(char),hipMemcpyHostToDevice); 
	}
    }
/*******************  END  *****************************/

/*******************  CUDA KERNEL LAUNCH  *************************************/
    // Each block will corresponds to 1 transaction
    // and the threads within a block correspond to items in that transaction
    // I purposely set them in these ways to avoid/miminize control divergence problems - Better Performance
    // This also makes coding slightly simpler (not much), btw! 
    dim3 dimGrid(ceil(sqrt(num_of_transactions)),ceil(sqrt(num_of_transactions)),1); 
    dim3 dimBlock(ceil(sqrt(num_of_itemsets)),ceil(sqrt(num_of_itemsets)),1);

    // Launch kernel to find the transpose of db
    cudaTranspose <<< dimGrid, dimBlock >>> (db_gpuPitchPtr, db_gpuDim, db_tran_gpuPitchPtr);
    hipDeviceSynchronize();

    // Launch kernel to find all the entries of the Equivalent Classes of upto length 2 and sort them simultaneously
    cudaEC_len2 <<< dimGrid, dimBlock >>> (db_gpuPitchPtr, db_gpuDim, db_tran_gpuPitchPtr, EC_len2_gpuPitchPtr, output_element_len2_gpuPitchPtr, output_support_confidence_len2_gpuPitchPtr, EC_size);
    hipDeviceSynchronize();
/*******************  END  *****************************/

/*******************  CUDA DEVICE TO HOST COPY  ***********************************************/
    char *db_tran_ptr = (char*)db_tran_gpuPitchPtr.ptr;
    size_t db_tran_pitch = db_tran_gpuPitchPtr.pitch;
    size_t db_tran_slicePitch = db_tran_pitch * num_of_itemsets * num_of_transactions;
    char *db_tran_current_slice;
    char *db_tran_element;

    for (i = 0; i < num_of_itemsets*num_of_transactions; i++)  {
        for (j = 0; j < num_of_transactions+1; j++)  {
            db_tran_current_slice = db_tran_ptr + j * db_tran_slicePitch;
            db_tran_element = (char *)(db_tran_current_slice + i * db_tran_pitch);
            hipMemcpy(db_tran[i][j],db_tran_element,100*sizeof(char),hipMemcpyDeviceToHost);
        }
    }

    printf("++++++++++  Transpose of the Database  ++++++++++\n");
    for (i = 0; i < num_of_itemsets*num_of_transactions; i++)  {
	if (strcmp(db_tran[i][0],"") == 0)  continue;
	printf("%s: ",db_tran[i][0]);
        for (j = 1; j < num_of_transactions+1; j++)  {
            printf("-%s-",db_tran[i][j]);
        }
        printf("\n");
    }
/*************  Redundant  *************
    char *EC_len2_ptr = (char *)EC_len2_gpuPitchPtr.ptr;
    size_t EC_len2_pitch = EC_len2_gpuPitchPtr.pitch;
    size_t EC_len2_slicePitch = EC_len2_pitch * num_of_transactions * EC_size;
    char *EC_len2_slice;
    char *EC_len2_element;

    for (i = 0; i < num_of_transactions*EC_size; i++)  {
	for (j = 0; j < (num_of_transactions+2)*27; j++)  {
	    EC_len2_slice = EC_len2_ptr + i * EC_len2_pitch;
	    EC_len2_element = (char *)(EC_len2_slice + j * EC_len2_slicePitch);
	    hipMemcpy(EC_len2[i][j],EC_len2_element,100*sizeof(char),hipMemcpyDeviceToHost);
	}
    }

    int k;
    j = 0;
    printf("++++++++++  Equivalent Class of upto length 2  ++++++++++\n");
    for (k = 0; k < 27; k++)  {
        for (i = 0; i < num_of_transactions*EC_size; i++)  {
            if (strcmp(EC_len2[i][(k*(num_of_transactions+2))+0],"") == 0)  continue;
	    printf("{%s,%s}: ",EC_len2[i][(k*(num_of_transactions+2))+0],EC_len2[i][(k*(num_of_transactions+2))+1]);
            for (j = 2; j < num_of_transactions+2; j++)  {
                printf("-%s-",EC_len2[i][(k*(num_of_transactions+2))+j]);
            }
            printf("\n");
	}
    }
*******************  End  *******************/
    char *op_len2_ptr = (char *)output_element_len2_gpuPitchPtr.ptr;
    size_t op_len2_pitch = output_element_len2_gpuPitchPtr.pitch;
    size_t op_len2_slicePitch = op_len2_pitch * num_of_transactions * EC_size * 2;
    char *op_len2_slice;
    char *op_len2_element;

    for (i = 0; i < num_of_transactions*EC_size*2; i++)  {
        for (j = 0; j < 2; j++)  {
	    op_len2_slice = op_len2_ptr + i * op_len2_pitch;
	    op_len2_element = (char *)(op_len2_slice + j * op_len2_slicePitch);
	    hipMemcpy(output_element_len2[i][j],op_len2_element,100*sizeof(char),hipMemcpyDeviceToHost);
	}
    }

    float *op_sc_len2_ptr = (float *)output_support_confidence_len2_gpuPitchPtr.ptr;
    size_t op_sc_len2_pitch = output_support_confidence_len2_gpuPitchPtr.pitch;
    size_t op_sc_len2_slicePitch = op_sc_len2_pitch * num_of_transactions * EC_size * 2;
    float *op_sc_len2_slice;
    float *op_sc_len2_element;

    float output_support_confidence_len2[num_of_transactions*EC_size*2][2];

    for (i = 0; i < num_of_transactions*EC_size*2; i++)  {
        for (j = 0; j < 2; j++)  {
            op_sc_len2_slice = op_sc_len2_ptr + i * op_sc_len2_pitch;
            op_sc_len2_element = (float *)(op_sc_len2_slice + j * op_sc_len2_slicePitch);
            hipMemcpy(&output_support_confidence_len2[i][j],&op_sc_len2_element[0],sizeof(float),hipMemcpyDeviceToHost);
        }
    }

    j = 0;
    printf("++++++++++  Results upto length 2  ++++++++++\n");
    printf("{X} => {Y}:  support %%         confidence %% \n\n");
    for (i = 0; i < num_of_transactions*EC_size*2; i++)  {
        if (strcmp(output_element_len2[i][0],"") == 0)  continue;
        printf("{%s} => {%s}:  ",output_element_len2[i][0],output_element_len2[i][1]);
        for (j = 0; j < 2; j++)  {
                printf("%.4f %%        ",output_support_confidence_len2[i][j]);
        }
        printf("\n");
    }
/*******************  END  *****************************/
    printf("++++++++++  End  ++++++++++\n");

    hipFree(db_gpuPitchPtr.ptr);
    hipFree(db_tran_gpuPitchPtr.ptr);

//  free(db);
    return 0;
}

